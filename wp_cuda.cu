#include "hip/hip_runtime.h"
#include "wp_cuda.h"

hipDoubleComplex *dev_psi;
hipDoubleComplex *dev_psin1;
hipDoubleComplex *dev_psin2;
double          *dev_x;
double          *dev_dens;
double          *dev_Vx;
UNINT Ncores;

__global__ void propagate_psi(hipDoubleComplex *psin_grid,
                              hipDoubleComplex *psi_grid, double *Vx_grid,
                              int n_grid, double u_term, double dt){

    int ind = threadIdx.x + blockIdx.x * blockDim.x;
    hipDoubleComplex idt = make_hipDoubleComplex(0.0e0,-dt);
    hipDoubleComplex u2VC;
    hipDoubleComplex uC;
    hipDoubleComplex s1;
    hipDoubleComplex s2;
    hipDoubleComplex aux1;

    if (ind > 0 && ind < n_grid-1){
        u2VC = make_hipDoubleComplex(2*u_term+Vx_grid[ind],0.0e0);
        uC   = make_hipDoubleComplex(-u_term,0.0e0);
        s1   = hipCadd(psi_grid[ind-1], psi_grid[ind+1]);
        s1   = hipCmul(uC, s1);
        s2   = hipCmul(u2VC, psi_grid[ind]);
        aux1 = hipCadd(s1,s2);
        aux1 = hipCmul(idt,aux1);
        psin_grid[ind] = aux1;
    }

    return;

}

__global__ void update_psi(hipDoubleComplex *psi_grid,
                           hipDoubleComplex *psin1_grid,
                           hipDoubleComplex *psin2_grid,
                           int n_grid){

    int ind = threadIdx.x + blockIdx.x * blockDim.x;
    hipDoubleComplex aux1;
    hipDoubleComplex aux2;
    hipDoubleComplex f_half= make_hipDoubleComplex(0.5e0,0.0e0);

    if (ind > 0 && ind < n_grid-1){
        aux1 = hipCmul(f_half, psin2_grid[ind]);
        aux2 = hipCadd(psin1_grid[ind],aux1);
        psi_grid[ind] = hipCadd(psi_grid[ind],aux2);
    }
    else if (ind == 0 || ind == n_grid-1){
        psi_grid[ind] = make_hipDoubleComplex(0.0e0,0.0e0);
    }

    return;
}

__global__ void calc_dens(hipDoubleComplex *psi_grid, double *dens_grid,
                          int n_grid){

    int    ind = threadIdx.x + blockIdx.x * blockDim.x;
    double vabs;

    if (ind > 0 && ind < n_grid){
        vabs            = hipCabs(psi_grid[ind]);
        dens_grid[ind] = pow(vabs,2.0);
    }

    return;
}

void init_cuda_subs(complex<double> *psi_grid, double *x_grid,
                    complex<double> *psin1_grid,
                    complex<double> *psin2_grid,
                    double *dens_grid,
                    double *Vx_grid, UNINT n_grid){

    double gaux = (double) n_grid;
    double taux = (double) Nthreads;

    Ncores = (UNINT) ceil(gaux/taux);

    hipMalloc((void**) &dev_psi   , n_grid * sizeof(hipDoubleComplex));
    hipMalloc((void**) &dev_psin1 , n_grid * sizeof(hipDoubleComplex));
    hipMalloc((void**) &dev_psin2 , n_grid * sizeof(hipDoubleComplex));
    hipMalloc((void**) &x_grid    , n_grid * sizeof(double));
    hipMalloc((void**) &dev_dens  , n_grid * sizeof(double));
    hipMalloc((void**) &dev_Vx    , n_grid * sizeof(double));

    hipMemcpy(dev_psi, psi_grid, n_grid * sizeof(hipDoubleComplex),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_psin1, psin1_grid, n_grid * sizeof(hipDoubleComplex),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_psin2, psin2_grid, n_grid * sizeof(hipDoubleComplex),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_x, x_grid, n_grid * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_dens, dens_grid, n_grid * sizeof(double),
               hipMemcpyHostToDevice);
    hipMemcpy(dev_Vx, Vx_grid, n_grid * sizeof(double),
               hipMemcpyHostToDevice);

    return;

}

void propagate_cuda(UNINT n_grid, double mass, double dx, double dt){

    double u_term = 0.5*(1.0/(mass*pow(dx,2.0)));

    propagate_psi<<<Ncores, Nthreads>>>(dev_psin1, dev_psi, dev_Vx, n_grid,
                                        u_term, dt);
    propagate_psi<<<Ncores, Nthreads>>>(dev_psin2, dev_psin1, dev_Vx, n_grid,
                                        u_term, dt);
    update_psi<<<Ncores, Nthreads>>>(dev_psi, dev_psin1,dev_psin2, n_grid);

    return;
}

void get_cuda_psi2(complex<double> *psi_grid, double *dens_grid, int n_grid){

    calc_dens<<<Ncores, Nthreads>>>(dev_psi, dev_dens, n_grid);

    hipMemcpy(dens_grid, dev_dens, n_grid*sizeof(double),
               hipMemcpyDeviceToHost);
    hipMemcpy(psi_grid, dev_psi, n_grid*sizeof(hipDoubleComplex),
               hipMemcpyDeviceToHost);
    return;
}

void free_cuda_memory(){

    hipFree(dev_psi);
    hipFree(dev_psin1);
    hipFree(dev_psin2);
    hipFree(dev_x);
    hipFree(dev_dens);
    hipFree(dev_Vx);

    return;
}
